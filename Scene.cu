#include "hip/hip_runtime.h"
#include "Scene.h"

//__device__ Scene* currentScene;
//__host__
__managed__ Scene* currentScene = 0; // TODO use __const__ memory, not changeable from gpu!

CPU_AND_GPU Scene* Scene::getCurrentScene() {
    return currentScene;
}

__managed__ ITMVoxelBlock* currentLocalVBA = 0;
__device__ void Scene::AllocateVB::allocate(VoxelBlockPos pos, int sequenceId) {
    assert(currentLocalVBA);

    currentLocalVBA[sequenceId].reinit(pos);
}

void Scene::setCurrentScene(Scene* s) {
    hipDeviceSynchronize(); // want to write managed currentScene 
    currentScene = s;
}

Scene::Scene() {
    initCoordinateSystems();
    voxelBlockHash = new HashMap<Z3Hasher, AllocateVB>(SDF_EXCESS_LIST_SIZE);
    cudaSafeCall(hipMalloc(&localVBA, sizeof(ITMVoxelBlock) *SDF_LOCAL_BLOCK_NUM));
}

Scene::~Scene() {
    delete voxelBlockHash;
    hipFree(localVBA);
}

void Scene::performAllocations() {
    currentLocalVBA = localVBA;
    voxelBlockHash->performAllocations();
}

static GPU_ONLY inline VoxelBlockPos pointToVoxelBlockPos(
    const THREADPTR(Vector3i) & point //!< [in] in voxel coordinates
    ) {
    // "The 3D voxel block location is obtained by dividing the voxel coordinates with the block size along each axis."
    VoxelBlockPos blockPos;
    // if SDF_BLOCK_SIZE == 8, then -3 should go to block -1, so we need to adjust negative values 
    // (C's quotient-remainder division gives -3/8 == 0)
    blockPos.x = ((point.x < 0) ? point.x - SDF_BLOCK_SIZE + 1 : point.x) / SDF_BLOCK_SIZE;
    blockPos.y = ((point.y < 0) ? point.y - SDF_BLOCK_SIZE + 1 : point.y) / SDF_BLOCK_SIZE;
    blockPos.z = ((point.z < 0) ? point.z - SDF_BLOCK_SIZE + 1 : point.z) / SDF_BLOCK_SIZE;
    return blockPos;
}

GPU_ONLY ITMVoxel* Scene::getVoxel(Vector3i point) {
    VoxelBlockPos blockPos = pointToVoxelBlockPos(point);

    ITMVoxelBlock* b = getVoxelBlock(blockPos);
    if (b == NULL) return NULL;

    Vector3i localPos = point - blockPos.toInt() * SDF_BLOCK_SIZE; // localized coordinate
    return b->getVoxel(localPos);
}

/// Returns NULL if the voxel block is not allocated
GPU_ONLY ITMVoxelBlock* Scene::getVoxelBlock(VoxelBlockPos pos) {
    int sequenceNumber = voxelBlockHash->getSequenceNumber(pos);
    if (sequenceNumber == 0) return NULL;
    return &localVBA[sequenceNumber];
}

GPU_ONLY void Scene::requestVoxelBlockAllocation(VoxelBlockPos pos) {
    voxelBlockHash->requestAllocation(pos);
}

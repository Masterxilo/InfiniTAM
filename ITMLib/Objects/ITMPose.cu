#include "hip/hip_runtime.h"
#include <math.h>
#include "ITMPose.h"

#include <stdio.h>


ITMPose::ITMPose(void) { this->SetFrom(0, 0, 0, 0, 0, 0); }

ITMPose::ITMPose(float tx, float ty, float tz, float rx, float ry, float rz) 
{ this->SetFrom(tx, ty, tz, rx, ry, rz); }
ITMPose::ITMPose(const float pose[6]) { this->SetFrom(pose); }
ITMPose::ITMPose(const Matrix4f & src) { this->SetM(src); }
ITMPose::ITMPose(const Vector6f & tangent) { this->SetFrom(tangent); }
ITMPose::ITMPose(const ITMPose & src) { this->SetFrom(&src); }

#ifndef M_SQRT1_2
#define M_SQRT1_2 0.707106781186547524401
#endif

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#ifndef M_PI_2
#define M_PI_2 1.5707963267948966192E0
#endif

void ITMPose::SetFrom(float tx, float ty, float tz, float rx, float ry, float rz)
{
	this->params.each.tx = tx;
	this->params.each.ty = ty;
	this->params.each.tz = tz;
	this->params.each.rx = rx;
	this->params.each.ry = ry;
	this->params.each.rz = rz;

	this->SetModelViewFromParams();
}

void ITMPose::SetFrom(const Vector3f &translation, const Vector3f &rotation)
{
	this->params.each.tx = translation.x;
	this->params.each.ty = translation.y;
	this->params.each.tz = translation.z;
	this->params.each.rx = rotation.x;
	this->params.each.ry = rotation.y;
	this->params.each.rz = rotation.z;

	this->SetModelViewFromParams();
}

void ITMPose::SetFrom(const Vector6f &tangent)
{
	this->params.each.tx = tangent[0];
	this->params.each.ty = tangent[1];
	this->params.each.tz = tangent[2];
	this->params.each.rx = tangent[3];
	this->params.each.ry = tangent[4];
	this->params.each.rz = tangent[5];

	this->SetModelViewFromParams();
}

void ITMPose::SetFrom(const float pose[6])
{
	SetFrom(pose[0], pose[1], pose[2], pose[3], pose[4], pose[5]);
}

void ITMPose::SetFrom(const ITMPose *pose)
{
	this->params.each.tx = pose->params.each.tx;
	this->params.each.ty = pose->params.each.ty;
	this->params.each.tz = pose->params.each.tz;
	this->params.each.rx = pose->params.each.rx;
	this->params.each.ry = pose->params.each.ry;
	this->params.each.rz = pose->params.each.rz;

	M = pose->M;
}

void ITMPose::SetModelViewFromParams()
{
	// w is an "Euler vector", i.e. the vector "axis of rotation (u) * theta" (axis angle representation)
	const Vector3f w = params.r;
    const float theta_sq = dot(w,w), theta = sqrt(theta_sq);
	const float inv_theta = 1.0f / theta;

	const Vector3f t = params.t;

    float A, B, C;
	/*
	Limit for t approximating theta

	A = lim_{t -> theta} Sin[t]/t
	B = lim_{t -> theta} (1 - Cos[t])/t^2
	C = lim_{t -> theta} (1 - A)/t^2
	*/
    if (theta_sq < 1e-6f) // dont divide by very small or zero theta - use taylor series expansion of involved functions instead
    {
        A = 1     - theta_sq / 6 + theta_sq*theta_sq / 120; // Series[a, {t, 0, 4}]
        B = 1/2.f - theta_sq / 24;  //  Series[b, {t, 0, 2}]
		C = 1/6.f - theta_sq / 120; // Series[c, {t, 0, 2}]
    }
    else {
        A = sinf(theta) * inv_theta;
        B = (1.0f - cosf(theta)) * (inv_theta * inv_theta);
        C = (1.0f - A) * (inv_theta * inv_theta);
    }
    // TODO why isnt T = t?
	const Vector3f crossV = cross(w, t);
	const Vector3f cross2 = cross(w, crossV);
	const Vector3f T = t +  B * crossV + C * cross2;
	
	// w = t u, u \in S^2, t === theta
	// R = exp(w . L) = I + sin(t) (u . L) + (1 - cos(t)) (u . L)^2
	// u . L == [u]_x, the matrix computing the left cross product with u (u x *)
	// L = (L_x, L_y, L_z) the lie algebra basis
	// c.f. https://en.wikipedia.org/wiki/Rotation_group_SO(3)#Exponential_map
	Matrix3f R;
	const float wx2 = w.x * w.x, wy2 = w.y * w.y, wz2 = w.z * w.z;
	Rij(0, 0) = 1.0f - B*(wy2 + wz2);
	Rij(1, 1) = 1.0f - B*(wx2 + wz2);
	Rij(2, 2) = 1.0f - B*(wx2 + wy2);

	float a, b;
	a = A * w.z, b = B * (w.x * w.y);
	Rij(0, 1) = b - a;
	Rij(1, 0) = b + a;

	a = A * w.y, b = B * (w.x * w.z);
	Rij(0, 2) = b + a;
	Rij(2, 0) = b - a;

	a = A * w.x, b = B * (w.y * w.z);
	Rij(1, 2) = b - a;
	Rij(2, 1) = b + a;

	// Copy to M
	SetRPartOfM(R);
	M.setTranslate(T); 

	M.m[3 + 4*0] = 0.0f; M.m[3 + 4*1] = 0.0f; M.m[3 + 4*2] = 0.0f; M.m[3 + 4*3] = 1.0f;
}

void ITMPose::SetParamsFromModelView()
{
	// Compute this->params.r = resultRot;
	Vector3f resultRot;
	const Matrix3f R = GetR();

	const float cos_angle = (R.m00  + R.m11 + R.m22 - 1.0f) * 0.5f;
	resultRot.x = (Rij(2, 1) - Rij(1, 2)) * 0.5f;
	resultRot.y = (Rij(0, 2) - Rij(2, 0)) * 0.5f;
	resultRot.z = (Rij(1, 0) - Rij(0, 1)) * 0.5f;

	const float sin_angle_abs = length(resultRot);

	if (cos_angle > M_SQRT1_2)
	{
		if (sin_angle_abs) 
		{
			const float p = asinf(sin_angle_abs) / sin_angle_abs;
			resultRot *= p;
		}
	}
	else
	{
		if (cos_angle > -M_SQRT1_2)
		{
			const float p = acosf(cos_angle) / sin_angle_abs;
			resultRot *= p;
		}
		else
		{
			const float angle = (float)M_PI - asinf(sin_angle_abs);
			const float d0 = Rij(0, 0) - cos_angle;
			const float d1 = Rij(1, 1) - cos_angle;
			const float d2 = Rij(2, 2) - cos_angle;

			Vector3f r2;

			if(fabsf(d0) > fabsf(d1) && fabsf(d0) > fabsf(d2)) {
				r2.x = d0;
				r2.y = (Rij(1, 0) + Rij(0, 1)) * 0.5f;
				r2.z = (Rij(0, 2) + Rij(2, 0)) * 0.5f; 
			} else {
				if(fabsf(d1) > fabsf(d2)) {
					r2.x = (Rij(1, 0) + Rij(0, 1)) * 0.5f; 
					r2.y = d1; 
					r2.z = (Rij(2, 1) + Rij(1, 2)) * 0.5f; 
				}
				else {
					r2.x = (Rij(0, 2) + Rij(2, 0)) * 0.5f;
					r2.y = (Rij(2, 1) + Rij(1, 2)) * 0.5f; 
					r2.z = d2;
				}
			}

			if (dot(r2, resultRot) < 0.0f) { r2 *= -1.0f; }

			r2 = normalize(r2);

			resultRot = angle * r2; 
		}
	}

	this->params.r = resultRot;

	// Compute this->params.t = rottrans
	const Vector3f T = GetT();
	const float theta = length(resultRot);

    const float shtot = (theta > 0.00001f) ?
        sinf(theta * 0.5f) / theta :
        0.5f; // lim_{t -> theta} sin(t/2)/t, lim_{t -> 0} sin(t/2)/t = 0.5

	const ITMPose halfrotor(
		0.0f, 0.0f, 0.0f, 
		resultRot.x * -0.5f, resultRot.y * -0.5f, resultRot.z * -0.5f
		);

	Vector3f rottrans = halfrotor.GetR() * T;

    const float param = dot(T, resultRot) * 
        (
        (theta > 0.001f) ?
        (1 - 2 * shtot) / (theta * theta) :
        1 / 24.f // Series[(1 - 2*Sin[t/2]/t)/(t^2), {t, 0, 1}] = 1/24
        );
		
	rottrans -= resultRot * param;

	rottrans /= 2 * shtot;

	this->params.t = rottrans;
}

ITMPose ITMPose::exp(const Vector6f& tangent)
{
	return ITMPose(tangent);
}

void ITMPose::MultiplyWith(const ITMPose *pose)
{
	M = M * pose->M;
	this->SetParamsFromModelView();
}

Matrix3f ITMPose::GetR(void) const
{
    return M.GetR();
}

Vector3f ITMPose::GetT(void) const
{
	return M.getTranslate();
}

void ITMPose::GetParams(Vector3f &translation, Vector3f &rotation)
{
	translation.x = this->params.each.tx;
	translation.y = this->params.each.ty;
	translation.z = this->params.each.tz;

	rotation.x = this->params.each.rx;
	rotation.y = this->params.each.ry;
	rotation.z = this->params.each.rz;
}

void ITMPose::SetM(const Matrix4f & src)
{
	M = src;
	SetParamsFromModelView();
}

void ITMPose::SetR(const Matrix3f & R)
{
	SetRPartOfM(R);
	SetParamsFromModelView();
}

void ITMPose::SetT(const Vector3f & t)
{
	M.setTranslate(t);

	SetParamsFromModelView();
}

void ITMPose::SetRT(const Matrix3f & R, const Vector3f & t)
{
	SetRPartOfM(R);
	M.setTranslate(t);

	SetParamsFromModelView();
}

Matrix4f ITMPose::GetInvM(void) const
{
	Matrix4f ret;
	M.inv(ret);
	return ret;
}

void ITMPose::SetInvM(const Matrix4f & invM)
{
	invM.inv(M);
	SetParamsFromModelView();
}

void ITMPose::Coerce(void)
{
	SetParamsFromModelView();
	SetModelViewFromParams();
}


#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSceneReconstructionEngine_CUDA.h"
#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMSceneReconstructionEngine.h"
#include "../../../Objects/ITMRenderState_VH.h"


using namespace ITMLib::Engine;

// device functions

template<class TVoxel, bool stopIntegratingAtMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
    const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d,
    Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    Vector4f pt_model; int locId;

    locId = x + y * arrayInfo->size.x + z * arrayInfo->size.x * arrayInfo->size.y;

    if (stopIntegratingAtMaxW) if (voxelArray[locId].w_depth == maxW) return;
    //	if (approximateIntegration) if (voxelArray[locId].w_depth != 0) return;

    pt_model.x = (float)(x + arrayInfo->offset.x) * _voxelSize;
    pt_model.y = (float)(y + arrayInfo->offset.y) * _voxelSize;
    pt_model.z = (float)(z + arrayInfo->offset.z) * _voxelSize;
    pt_model.w = 1.0f;

    ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation, TVoxel>::compute(voxelArray[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

template<class TVoxel, bool stopIntegratingAtMaxW, bool approximateIntegration>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *visibleEntryIDs,
    const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d,
    Vector4f projParams_rgb, float voxelSize, float mu, int maxW)
{
    // one thread block for each voxel block
    Vector3i globalPos;
    int entryId = visibleEntryIDs[blockIdx.x];

    const ITMHashEntry &currentHashEntry = hashTable[entryId];

    if (currentHashEntry.ptr < 0) return;

    globalPos = currentHashEntry.pos.toInt() * SDF_BLOCK_SIZE;

    TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

    // one thread for each voxel
    int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;
    integrateVoxel(x, y, z,
        stopIntegratingAtMaxW, globalPos, localVoxelBlock, voxelSize,
        M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords, const float *depth,
    Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
    float viewFrustum_max)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

    buildHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
        projParams_d, mu, _imgSize, _voxelSize, hashTable, viewFrustum_min, viewFrustum_max);
}

__global__ void setToType3(uchar *entriesVisibleType, int *visibleEntryIDs, int noVisibleEntries)
{
    int entryId = threadIdx.x + blockIdx.x * blockDim.x;
    if (entryId > noVisibleEntries - 1) return;
    entriesVisibleType[visibleEntryIDs[entryId]] = 3;
}

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
    AllocationTempData *allocData, uchar *entriesAllocType, uchar *entriesVisibleType, Vector4s *blockCoords)
{
    int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (targetIdx > noTotalEntries - 1) return;

    allocateVoxelBlock(targetIdx,
        voxelAllocationList,
        excessAllocationList,
        hashTable,
        allocData,

        entriesAllocType,
        entriesVisibleType,
        blockCoords);
}

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
    AllocationTempData *allocData, /*int *noAllocatedVoxelEntries,*/ uchar *entriesVisibleType)
{
    int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (targetIdx > noTotalEntries - 1) return;

    reAllocateSwappedOutVoxelBlock(voxelAllocationList, targetIdx, entriesVisibleType, hashTable, allocData);
}

template<bool useSwapping>
__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashSwapState *swapStates, int noTotalEntries,
    int *visibleEntryIDs, AllocationTempData *allocData, uchar *entriesVisibleType,
    Matrix4f M_d, Vector4f projParams_d, Vector2i depthImgSize, float voxelSize)
{
    int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (targetIdx > noTotalEntries - 1) return;

    __shared__ bool shouldPrefix;
    shouldPrefix = false;
    __syncthreads();

    bool visible = visibilityTestIfNeeded(
        targetIdx, entriesVisibleType, useSwapping, hashTable, swapStates,
        M_d, projParams_d, depthImgSize, voxelSize
        );

    if (visible) shouldPrefix = true;

    __syncthreads();

    if (shouldPrefix)
    {
        int offset = computePrefixSum_device<int>(visible, &allocData->noVisibleEntries, blockDim.x * blockDim.y, threadIdx.x);
        if (offset != -1) visibleEntryIDs[offset] = targetIdx;
    }
}

// host methods

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipMalloc((void**)&allocationTempData_device, sizeof(AllocationTempData)));
	ITMSafeCall(hipHostMalloc((void**)&allocationTempData_host, sizeof(AllocationTempData)));

	int noTotalEntries = ITMVoxelBlockHash::noTotalEntries;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector4s)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipHostFree(allocationTempData_host));
	ITMSafeCall(hipFree(allocationTempData_device));
	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ResetScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

    // Reset all voxels in all voxel blocks
	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);

    // Reset voxel allocation list
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;

    // Reset hash entries
    ITMHashEntry tmpEntry = ITMHashEntry::createIllegalEntry();
	ITMHashEntry *hashEntry_ptr = scene->index.GetEntries();
	memsetKernel<ITMHashEntry>(hashEntry_ptr, tmpEntry, scene->index.noTotalEntries);

    // Reset excess allocation list
	int *excessList_ptr = scene->index.GetExcessAllocationList();
	fillArrayKernel<int>(excessList_ptr, SDF_EXCESS_LIST_SIZE);
	scene->index.SetLastFreeExcessListId(SDF_EXCESS_LIST_SIZE - 1);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view, 
	const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->GetM(); M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	ITMHashEntry *hashTable = scene->index.GetEntries();


    // [swapping[
	ITMHashSwapState *swapStates = scene->useSwapping ? scene->globalCache->GetSwapStates(true) : 0;
    bool useSwapping = scene->useSwapping;
    // ]swapping]

	int noTotalEntries = scene->index.noTotalEntries;

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	dim3 cudaBlockSizeVS(256, 1);
	dim3 gridSizeVS((int)ceil((float)renderState_vh->noVisibleEntries / (float)cudaBlockSizeVS.x));

	float oneOverVoxelSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);

	AllocationTempData *tempData = (AllocationTempData*)allocationTempData_host;
	tempData->noAllocatedVoxelEntries = scene->localVBA.lastFreeBlockId;
	tempData->noAllocatedExcessEntries = scene->index.GetLastFreeExcessListId();
	tempData->noVisibleEntries = 0;
	ITMSafeCall(hipMemcpyAsync(allocationTempData_device, tempData, sizeof(AllocationTempData), hipMemcpyHostToDevice));

	ITMSafeCall(hipMemsetAsync(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));

	if (gridSizeVS.x > 0) setToType3 << <gridSizeVS, cudaBlockSizeVS >> > (entriesVisibleType, visibleEntryIDs, renderState_vh->noVisibleEntries);

	buildHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType, 
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max);

	if (onlyUpdateVisibleList) useSwapping = false;
	if (!onlyUpdateVisibleList)
	{
		allocateVoxelBlocksList_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, excessAllocationList, hashTable,
			noTotalEntries, (AllocationTempData*)allocationTempData_device, entriesAllocType_device, entriesVisibleType,
			blockCoords_device);
	}

	if (useSwapping)
		buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize);
	else
		buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, swapStates, noTotalEntries, visibleEntryIDs,
			(AllocationTempData*)allocationTempData_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize);

	if (useSwapping)
	{
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries, 
			(AllocationTempData*)allocationTempData_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(tempData, allocationTempData_device, sizeof(AllocationTempData), hipMemcpyDeviceToHost));
	renderState_vh->noVisibleEntries = tempData->noVisibleEntries;
	scene->localVBA.lastFreeBlockId = tempData->noAllocatedVoxelEntries;
	scene->index.SetLastFreeExcessListId(tempData->noAllocatedExcessEntries);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::IntegrateIntoScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(renderState_vh->noVisibleEntries);

#define integrateIntoScene_d(stopIntegratingAtMaxW, approximateIntegration) \
    integrateIntoScene_device<TVoxel, stopIntegratingAtMaxW, approximateIntegration> << <gridSize, cudaBlockSize >> >(\
    localVBA, hashTable, visibleEntryIDs,\
        rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW)

	if (scene->sceneParams->stopIntegratingAtMaxW)
		if (trackingState->requiresFullRendering)
            integrateIntoScene_d(true, false);
		else
            integrateIntoScene_d(true, true);
	else
		if (trackingState->requiresFullRendering)
            integrateIntoScene_d(false, false);
		else
            integrateIntoScene_d(false, true);
}

// plain voxel array

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel,ITMPlainVoxelArray>::ResetScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene)
{
	int numBlocks = scene->index.getNumAllocatedVoxelBlocks();
	int blockSize = scene->index.getVoxelBlockSize();

	TVoxel *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
	memsetKernel<TVoxel>(voxelBlocks_ptr, TVoxel(), numBlocks * blockSize);
	int *vbaAllocationList_ptr = scene->localVBA.GetAllocationList();
	fillArrayKernel<int>(vbaAllocationList_ptr, numBlocks);
	scene->localVBA.lastFreeBlockId = numBlocks - 1;
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState, bool onlyUpdateVisibleList)
{
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::IntegrateIntoScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	M_d = trackingState->pose_d->GetM();
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo = scene->index.getIndexData();

	dim3 cudaBlockSize(8, 8, 8);
	dim3 gridSize(scene->index.getVolumeSize().x / cudaBlockSize.x, scene->index.getVolumeSize().y / cudaBlockSize.y, scene->index.getVolumeSize().z / cudaBlockSize.z);

	if (scene->sceneParams->stopIntegratingAtMaxW) {
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, true, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, true, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
	else
	{
		if (trackingState->requiresFullRendering)
			integrateIntoScene_device < TVoxel, false, false> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
		else
			integrateIntoScene_device < TVoxel, false, true> << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
				rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
	}
}



template class ITMLib::Engine::ITMSceneReconstructionEngine_CUDA<ITMVoxel, ITMVoxelIndex>;


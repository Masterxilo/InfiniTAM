#include "hip/hip_runtime.h"
﻿#include "ITMVisualisationEngine.h"
#include "ITMPixelUtils.h"
#include "ITMCUDAUtils.h"

#include "ITMRepresentationAccess.h"
#include "ITMLibDefines.h"
#include "ITMSceneReconstructionEngine.h"

using namespace ITMLib::Engine;



#ifndef FAR_AWAY
#define FAR_AWAY 999999.9f
#endif

#ifndef VERY_CLOSE
#define VERY_CLOSE 0.05f
#endif

static const CONSTPTR(int) renderingBlockSizeX = 16;
static const CONSTPTR(int) renderingBlockSizeY = 16;

static const CONSTPTR(int) MAX_RENDERING_BLOCKS = 65536 * 4;
//static const int MAX_RENDERING_BLOCKS = 16384;
static const CONSTPTR(int) minmaximg_subsample = 8;

/**
Project visible blocks into the desired image.

Compute the bounding box (upperLeft, lowerRight, zRange) of the
projection of all eight corners in image space and store the minimum
and maximum Z coordinates of the block in the camera coordinate
system
*/
CPU_AND_GPU inline bool ProjectSingleBlock(
    const THREADPTR(Vector3s) & blockPos,
    const THREADPTR(Matrix4f) & pose,
    const THREADPTR(Vector4f) & intrinsics,
    const THREADPTR(Vector2i) & imgSize,
    float voxelSize,
    THREADPTR(Vector2i) & upperLeft, //!< [out]
    THREADPTR(Vector2i) & lowerRight,  //!< [out]
    THREADPTR(Vector2f) & zRange //!< [out]
    )
{
    const Vector2i minmaxImgSize = imgSize / minmaximg_subsample;
    upperLeft = minmaxImgSize;
    lowerRight = Vector2i(-1, -1);
    // zMin, zmax
    zRange = Vector2f(FAR_AWAY, VERY_CLOSE);

    // project all 8 corners down to 2D image
    for (int corner = 0; corner < 8; ++corner)
    {
        Vector3s tmp = blockPos;
        tmp.x += (corner & 1) ? 1 : 0;
        tmp.y += (corner & 2) ? 1 : 0;
        tmp.z += (corner & 4) ? 1 : 0;
        Vector4f pt3d(TO_FLOAT3(tmp) * (float)SDF_BLOCK_SIZE * voxelSize, 1.0f);
        pt3d = pose * pt3d;

        Vector2f pt2d;
        if (!projectNoBounds(intrinsics, pt3d, pt2d)) continue;
        pt2d /= minmaximg_subsample;

        // remember bounding box, zmin and zmax
        if (upperLeft.x > floor(pt2d.x)) upperLeft.x = (int)floor(pt2d.x);
        if (lowerRight.x < ceil(pt2d.x)) lowerRight.x = (int)ceil(pt2d.x);
        if (upperLeft.y > floor(pt2d.y)) upperLeft.y = (int)floor(pt2d.y);
        if (lowerRight.y < ceil(pt2d.y)) lowerRight.y = (int)ceil(pt2d.y);
        if (zRange.x > pt3d.z) zRange.x = pt3d.z;
        if (zRange.y < pt3d.z) zRange.y = pt3d.z;
    }

    // do some sanity checks and respect image bounds
    if (upperLeft.x < 0) upperLeft.x = 0;
    if (upperLeft.y < 0) upperLeft.y = 0;
    if (lowerRight.x >= minmaxImgSize.x) lowerRight.x = minmaxImgSize.x - 1;
    if (lowerRight.y >= minmaxImgSize.y) lowerRight.y = minmaxImgSize.y - 1;
    if (upperLeft.x > lowerRight.x) return false;
    if (upperLeft.y > lowerRight.y) return false;
    //if (zRange.y <= VERY_CLOSE) return false; never seems to happen
    if (zRange.x < VERY_CLOSE) zRange.x = VERY_CLOSE;
    if (zRange.y < VERY_CLOSE) return false;

    return true;
}

/**
Split image-depth space bounding box described by (upperLeft, lowerRight, zRange)
into (renderingBlockSizeX by renderingBlockSizeY) pixel (or less) RenderingBlocks of same zRange.

Store the resulting blocks into renderingBlockList,
incrementing the current position 'offset' in this list.
*/
CPU_AND_GPU inline void CreateRenderingBlocks(
    DEVICEPTR(RenderingBlock) *renderingBlockList, //!< [out]
    int offset, //!< [out]

    const THREADPTR(Vector2i) & upperLeft,
    const THREADPTR(Vector2i) & lowerRight,
    const THREADPTR(Vector2f) & zRange)
{
    for (int by = 0; by < ceil((float)(1 + lowerRight.y - upperLeft.y) / renderingBlockSizeY); ++by) {
        for (int bx = 0; bx < ceil((float)(1 + lowerRight.x - upperLeft.x) / renderingBlockSizeX); ++bx) {
            // End if list is full.
            if (offset >= MAX_RENDERING_BLOCKS) return;
            //for each rendering block: add it to the list
            DEVICEPTR(RenderingBlock) & b(renderingBlockList[offset++]);

            b.upperLeft.x = upperLeft.x + bx*renderingBlockSizeX;
            b.upperLeft.y = upperLeft.y + by*renderingBlockSizeY;

            // lowerRight corner
            b.lowerRight.x = upperLeft.x + (bx + 1)*renderingBlockSizeX - 1;
            b.lowerRight.y = upperLeft.y + (by + 1)*renderingBlockSizeY - 1;

            // Stay within image bounds (renderingBlockSizeX, renderingBlockSizeY) might not fit
            if (b.lowerRight.x>lowerRight.x) b.lowerRight.x = lowerRight.x;
            if (b.lowerRight.y>lowerRight.y) b.lowerRight.y = lowerRight.y;

            b.zRange = zRange;
        }
    }
}

/// \param x,y [in] camera space pixel determining ray direction
/// \returns whether any intersection was found
CPU_AND_GPU inline bool castRay(
    DEVICEPTR(Vector4f) &pt_out, //!< [out] the intersection point. w is 1 for a valid point, 0 for no intersection; in voxel-fractional-world-coordinates

    const int x, const int y,
    const CONSTPTR(ITMVoxelBlock) *voxelData,
    const CONSTPTR(typename ITMVoxelBlockHash::IndexData) *voxelIndex,
    const Matrix4f invM, //!< camera-to-world transform
    const Vector4f invProjParams, //!< camera-to-world transform
    const float oneOverVoxelSize,
    const float mu,
    const CONSTPTR(Vector2f) & viewFrustum_minmax)
{
    Vector4f pt_camera_f;
    Vector3f pt_block_s, pt_block_e;

    float totalLength;


    // Starting point
    pt_camera_f = depthTo3DInvProjParams(invProjParams, x, y, viewFrustum_minmax.x);
    // Lengths given in voxel-fractional-coordinates (such that one voxel has size 1)
    totalLength = length(TO_VECTOR3(pt_camera_f)) * oneOverVoxelSize;
    // in voxel-fractional-world-coordinates (such that one voxel has size 1)
    pt_block_s = TO_VECTOR3(invM * pt_camera_f) * oneOverVoxelSize;

    // End point
    pt_camera_f = depthTo3DInvProjParams(invProjParams, x, y, viewFrustum_minmax.y);
    const float totalLengthMax = length(TO_VECTOR3(pt_camera_f)) * oneOverVoxelSize;
    pt_block_e = TO_VECTOR3(invM * pt_camera_f) * oneOverVoxelSize;


    // Raymarching
    const Vector3f rayDirection = normalize(pt_block_e - pt_block_s);
    Vector3f pt_result = pt_block_s; // Current position in voxel-fractional-world-coordinates
    const float stepScale = mu * oneOverVoxelSize;
    typename ITMVoxelBlockHash::IndexCache cache;
    float sdfValue = 1.0f;
    bool hash_found;
    float stepLength;
    while (totalLength < totalLengthMax) {
        // D(X)
        sdfValue = readFromSDF_float_uninterpolated(voxelData, voxelIndex, pt_result, hash_found, cache);

        if (!hash_found) {
            //  First we try to find an allocated voxel block, and the length of the steps we take is determined by the block size
            stepLength = SDF_BLOCK_SIZE;
        }
        else {
            // If we found an allocated block, 
            // [Once we are inside the truncation band], the values from the SDF give us conservative step lengths.

            // using trilinear interpolation only if we have read values in the range −0.5 ≤ D(X) ≤ 0.1
            if ((sdfValue <= 0.1f) && (sdfValue >= -0.5f)) {
                sdfValue = readFromSDF_float_interpolated(voxelData, voxelIndex, pt_result, hash_found, cache);
            }
            // once we read a negative value from the SDF, we found the intersection with the surface.
            if (sdfValue <= 0.0f) break;

            stepLength = MAX(
                sdfValue * stepScale,
                1.0f // if we are outside the truncation band µ, our step size is determined by the truncation band 
                // (note that the distance is normalized to lie in [-1,1] within the truncation band)
                );
        }

        pt_result += stepLength * rayDirection;
        totalLength += stepLength;
    }

    bool pt_found;
    //  If the T - SDF value is negative after such a trilinear interpolation, the surface
    //  has indeed been found and we terminate the ray, performing one last
    //  trilinear interpolation step for a smoother appearance.
    if (sdfValue <= 0.0f)
    {
        // Refine position
        stepLength = sdfValue * stepScale;
        pt_result += stepLength * rayDirection;

        // Read again
        sdfValue = readFromSDF_float_interpolated(voxelData, voxelIndex, pt_result, hash_found, cache);
        // Refine position
        stepLength = sdfValue * stepScale;
        pt_result += stepLength * rayDirection;

        pt_found = true;
    }
    else pt_found = false;

    pt_out = Vector4f(pt_result, (pt_found) ? 1.0f : 0.0f);

    return pt_found;
}

/// Compute normal in the distance field via the gradient.
/// c.f. computeSingleNormalFromSDF
CPU_AND_GPU inline void computeNormalAndAngle(
    THREADPTR(bool) & foundPoint, //!< in,out
    const THREADPTR(Vector3f) & point,
    const CONSTPTR(ITMVoxelBlock) *voxelBlockData,
    const CONSTPTR(typename ITMVoxelBlockHash::IndexData) *indexData,
    const THREADPTR(Vector3f) & lightSource,
    THREADPTR(Vector3f) & outNormal,
    THREADPTR(float) & angle //!< outNormal . lightSource
    )
{
    if (!foundPoint) return;

    outNormal = normalize(computeSingleNormalFromSDF(voxelBlockData, indexData, point));

    angle = dot(outNormal, lightSource);
    if (!(angle > 0.0)) foundPoint = false;
}

/**
Computing the surface normal in image space given raycasted image.

In image space, since the normals are computed on a regular grid,
there are only 4 uninterpolated read operations followed by a cross-product.

\returns normal_out[idx].w = sigmaZ_out[idx] = -1 on error where idx = x + y * imgDims.x
*/
template <bool useSmoothing>
CPU_AND_GPU inline void computeNormalAndAngle(
    THREADPTR(bool) & foundPoint, //!< in,out. Set to false when the normal cannot be computed
    const THREADPTR(int) &x, const THREADPTR(int) &y,
    const CONSTPTR(Vector4f) *pointsRay,
    const THREADPTR(Vector3f) & lightSource,
    const THREADPTR(float) &voxelSize,
    const THREADPTR(Vector2i) &imgSize,
    THREADPTR(Vector3f) & outNormal,
    THREADPTR(float) & angle //!< outNormal . lightSource
    )
{
    if (!foundPoint) return;

    // Lookup world coordinates of points surrounding (x,y)
    // and compute forward difference vectors
    Vector4f xp1_y, xm1_y, x_yp1, x_ym1;
    Vector4f diff_x(0.0f, 0.0f, 0.0f, 0.0f), diff_y(0.0f, 0.0f, 0.0f, 0.0f);

    // If useSmoothing, use positions 2 away
    int extraDelta = useSmoothing ? 1 : 0;

#define d(x) (x + extraDelta)

    if (y <= d(1) || y >= imgSize.y - d(2) || x <= d(1) || x >= imgSize.x - d(2)) { foundPoint = false; return; }

#define lookupNeighbors() \
    xp1_y = sampleNearest(pointsRay, x + d(1), y, imgSize);\
    x_yp1 = sampleNearest(pointsRay, x, y + d(1), imgSize);\
    xm1_y = sampleNearest(pointsRay, x - d(1), y, imgSize);\
    x_ym1 = sampleNearest(pointsRay, x, y - d(1), imgSize);\
    diff_x = xp1_y - xm1_y;\
    diff_y = x_yp1 - x_ym1;

    lookupNeighbors();

#define isAnyPointIllegal() (xp1_y.w <= 0 || x_yp1.w <= 0 || xm1_y.w <= 0 || x_ym1.w <= 0)

    float length_diff = MAX(length2(diff_x.toVector3()), length2(diff_y.toVector3()));
    bool lengthDiffTooLarge = (length_diff * voxelSize * voxelSize > (0.15f * 0.15f));

    if (isAnyPointIllegal() || (lengthDiffTooLarge && useSmoothing)) {
        if (!useSmoothing) { foundPoint = false; return; }

        // In case we used smoothing, try again without extra delta 
        extraDelta = 0;
        lookupNeighbors();

        if (isAnyPointIllegal()){ foundPoint = false; return; }
    }

#undef d
#undef isAnyPointIllegal
#undef lookupNeighbors

    // TODO why the extra minus?
    outNormal = normalize(-cross(diff_x.toVector3(), diff_y.toVector3()));

    angle = dot(outNormal, lightSource);
    if (!(angle > 0.0)) foundPoint = false;
}




#define DRAWFUNCTIONPARAMS \
DEVICEPTR(Vector4u) & dest,\
const CONSTPTR(Vector3f) & point, /* in voxel-fractional world coordinates, comes from raycastResult*/\
const CONSTPTR(ITMVoxelBlock) *voxelBlockData, \
const CONSTPTR(typename ITMVoxelBlockHash::IndexData) *indexData,\
const THREADPTR(Vector3f) & normal_obj,\
const THREADPTR(float) & angle

// PIXEL SHADERS
// " Finally a coloured or shaded rendering of the surface is trivially computed, as desired for the visualisation."
CPU_AND_GPU inline void drawPixelGrey(DRAWFUNCTIONPARAMS)
{
    float outRes = (0.8f * angle + 0.2f) * 255.0f;
    dest = Vector4u((uchar)outRes);
}

CPU_AND_GPU inline void drawPixelNormal(DRAWFUNCTIONPARAMS)
{
    dest.r = (uchar)((0.3f + (-normal_obj.r + 1.0f)*0.35f)*255.0f);
    dest.g = (uchar)((0.3f + (-normal_obj.g + 1.0f)*0.35f)*255.0f);
    dest.b = (uchar)((0.3f + (-normal_obj.b + 1.0f)*0.35f)*255.0f);
}

CPU_AND_GPU inline void drawPixelColour(DRAWFUNCTIONPARAMS)
{
    Vector3f clr = readFromSDF_color4u_interpolated(voxelBlockData, indexData, point);
    dest = Vector4u(TO_UCHAR3(clr), 255); 
}

#define PROCESS_AND_DRAW_PIXEL(PROCESSFUNCTION, DRAWFUNCTION) \
CPU_AND_GPU inline void PROCESSFUNCTION(DEVICEPTR(Vector4u) &outRendering, const CONSTPTR(Vector3f) & point,\
    bool foundPoint, const CONSTPTR(ITMVoxelBlock) *voxelData, const CONSTPTR(typename ITMVoxelBlockHash::IndexData) *voxelIndex,\
	Vector3f lightSource) {\
	Vector3f outNormal;\
	float angle;\
    computeNormalAndAngle(foundPoint, point, voxelData, voxelIndex, lightSource, outNormal, angle);\
    if (foundPoint) DRAWFUNCTION(outRendering, point, voxelData, voxelIndex, outNormal, angle);\
    	else outRendering = Vector4u((uchar)0);\
}

PROCESS_AND_DRAW_PIXEL(processPixelColour, drawPixelColour)
PROCESS_AND_DRAW_PIXEL(processPixelGrey, drawPixelGrey)
PROCESS_AND_DRAW_PIXEL(processPixelNormal, drawPixelNormal)


CPU_AND_GPU inline void processPixelICPPost(
const float angle,
const Vector3f outNormal,
DEVICEPTR(Vector4f) &pointsMap, //<! [out] trackingState->pointCloud->locations (world space conversion of point)
DEVICEPTR(Vector4f) &normalsMap,
const THREADPTR(Vector3f) & point, //<! [in] renderState->raycastResult, in voxel-fractional-world-coordinates!
const bool foundPoint,
const float voxelSize)
{

    if (!foundPoint)
    {
        pointsMap = normalsMap = IllegalColor<Vector4f>::make();
        return;
    }

    pointsMap = Vector4f(point * voxelSize, 1);
    normalsMap = Vector4f(outNormal, 0);
}

/**
Produces a shaded image (outRendering) and a point cloud for e.g. tracking.
Uses image space normals.
*/
/// \param useSmoothing whether to compute normals by forward differences two pixels away (true) or just one pixel away (false)
template<bool useSmoothing>
CPU_AND_GPU inline void processPixelICP(
    DEVICEPTR(Vector4f) *const pointsMap, //!< [out] receives output points in world coordinates
    DEVICEPTR(Vector4f) *const normalsMap,

    const CONSTPTR(Vector4f) *pointsRay, //!< [in] points in voxel-fractional-world-coordinates (renderState->raycastResult)
    const THREADPTR(Vector2i) &imgSize,
    const THREADPTR(int) &x,
    const THREADPTR(int) &y,
    const float voxelSize,
    const THREADPTR(Vector3f) &lightSource)
{
    Vector3f outNormal;
    float angle;

    int locId = pixelLocId(x, y, imgSize);
    Vector4f point = pointsRay[locId];

    bool foundPoint = point.w > 0.0f;

    computeNormalAndAngle<useSmoothing>(foundPoint, x, y, pointsRay, lightSource, voxelSize, imgSize, outNormal, angle);

    processPixelICPPost(
        angle, outNormal,
        pointsMap[locId],
        normalsMap[locId],
        point.toVector3(),
        foundPoint,
        voxelSize);
}

/// as val goes from x0 to x1, output goes from y0 to y1 linearly
inline float interpolate(float val, float y0, float x0, float y1, float x1) {
	return (val - x0)*(y1 - y0) / (x1 - x0) + y0;
}

/**
1   ---
0__/   \___
where the angles are at
-.75, -.25, .25, .75
*/
inline float base(float val) {
	if (val <= -0.75f) return 0.0f;
	else if (val <= -0.25f) return interpolate(val, 0.0f, -0.75f, 1.0f, -0.25f);
	else if (val <= 0.25f) return 1.0f;
	else if (val <= 0.75f) return interpolate(val, 1.0f, 0.25f, 0.0f, 0.75f);
	else return 0.0;
}

void ITMVisualisationEngine::DepthToUchar4(ITMUChar4Image *dst, const ITMFloatImage *src)
{
    dst->Clear();
    Vector4u * const dest = dst->GetData(MEMORYDEVICE_CPU);
	float const * const source = src->GetData(MEMORYDEVICE_CPU);
	const int dataSize = static_cast<int>(dst->dataSize);

    // lims =  #@source & /@ {Min, Max}
	float lims[2];
	lims[0] = 100000.0f; lims[1] = -100000.0f;

	for (int idx = 0; idx < dataSize; idx++)
	{
		float sourceVal = source[idx];
		if (sourceVal > 0.0f) { lims[0] = MIN(lims[0], sourceVal); lims[1] = MAX(lims[1], sourceVal); }
	}
	if (lims[0] == lims[1]) return;

    // Rescaled rgb-converted depth
    const float scale = 1.0f / (lims[1] - lims[0]);
	for (int idx = 0; idx < dataSize; idx++)
	{
		float sourceVal = source[idx];

        if (sourceVal <= 0.0f) continue;
		sourceVal = (sourceVal - lims[0]) * scale;

        dest[idx].r = (uchar)(base(sourceVal - 0.5f) * 255.0f); // shows the range 0 to 1.25
		dest[idx].g = (uchar)(base(sourceVal) * 255.0f); // shows the range 0 to .75
		dest[idx].b = (uchar)(base(sourceVal + 0.5f) * 255.0f); // shows the range 
		dest[idx].a = 255;
	}
}


inline dim3 getGridSize(dim3 taskSize, dim3 blockSize)
{
    return dim3((taskSize.x + blockSize.x - 1) / blockSize.x, (taskSize.y + blockSize.y - 1) / blockSize.y, (taskSize.z + blockSize.z - 1) / blockSize.z);
}

inline dim3 getGridSize(Vector2i taskSize, dim3 blockSize) { return getGridSize(dim3(taskSize.x, taskSize.y), blockSize); }

//device implementations

KERNEL projectAndSplitBlocks_device(
    const ITMHashEntry * const hashEntries,
    const ITMVoxelBlock * const localVBA,
    const Matrix4f pose_M,
    const Vector4f intrinsics,
    const Vector2i imgSize,
    const float voxelSize,
    RenderingBlock *renderingBlocks, //!< [out]
    uint *noTotalBlocks //!< [out]
    )
{
    Vector2i upperLeft, lowerRight;
    Vector2f zRange;
    bool validProjection = false;

    const int in_offset = threadIdx.x + blockDim.x * blockIdx.x;

    // ignoring visible list:
    VoxelBlockPos pos = localVBA[in_offset].pos;
    if (pos != INVALID_VOXEL_BLOCK_POS)
        // Shared:
        validProjection = ProjectSingleBlock(
            pos,//blockData.pos, 
            pose_M, intrinsics, imgSize, voxelSize, upperLeft, lowerRight, zRange);

    Vector2i requiredRenderingBlocks(ceilf((float)(lowerRight.x - upperLeft.x + 1) / renderingBlockSizeX),
        ceilf((float)(lowerRight.y - upperLeft.y + 1) / renderingBlockSizeY));

    size_t requiredNumBlocks = requiredRenderingBlocks.x * requiredRenderingBlocks.y;
    if (!validProjection) requiredNumBlocks = 0;

    int out_offset = computePrefixSum_device<uint>(requiredNumBlocks, noTotalBlocks, blockDim.x, threadIdx.x);
    if (!validProjection) return;
    if ((out_offset == -1) || (out_offset + requiredNumBlocks > MAX_RENDERING_BLOCKS)) return;

    CreateRenderingBlocks(renderingBlocks, out_offset, upperLeft, lowerRight, zRange);
}

KERNEL fillBlocks_device(const uint *noTotalBlocks, const RenderingBlock *renderingBlocks,
    Vector2i imgSize,
    Vector2f *minmaxData //!< [out]
    )
{
    int x = threadIdx.x;
    int y = threadIdx.y;
    int block = blockIdx.x * 4 + blockIdx.y;
    if (block >= *noTotalBlocks) return;

    const RenderingBlock & b(renderingBlocks[block]);
    int xpos = b.upperLeft.x + x;
    if (xpos > b.lowerRight.x) return;
    int ypos = b.upperLeft.y + y;
    if (ypos > b.lowerRight.y) return;

    Vector2f & pixel(minmaxData[xpos + ypos*imgSize.x]);
    atomicMin(&pixel.x, b.zRange.x); atomicMax(&pixel.y, b.zRange.y);
}

KERNEL genericRaycast_device(Vector4f *out_ptsRay, const ITMVoxelBlock *voxelData, const typename ITMVoxelBlockHash::IndexData *voxelIndex,
    Vector2i imgSize, Matrix4f invM, Vector4f invProjParams, float oneOverVoxelSize, const Vector2f *minmaximg, float mu)
{
    int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

    if (x >= imgSize.x || y >= imgSize.y) return;

    int locId = x + y * imgSize.x;
    int locId2 = (int)floor((float)x / minmaximg_subsample) + (int)floor((float)y / minmaximg_subsample) * imgSize.x;

    castRay(out_ptsRay[locId], x, y, voxelData, voxelIndex, invM, invProjParams, oneOverVoxelSize, mu, minmaximg[locId2]);
}

KERNEL renderICP_device(Vector4f *pointsMap, Vector4f *normalsMap, const Vector4f *pointsRay,
    float voxelSize, Vector2i imgSize, Vector3f lightSource)
{
    int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

    if (x >= imgSize.x || y >= imgSize.y) return;

    processPixelICP<true>(pointsMap, normalsMap, pointsRay, imgSize, x, y, voxelSize, lightSource);
}

/*
renderGrey_device, processPixelGrey
renderColourFromNormal_device, processPixelNormal
renderColour_device, processPixelColour
*/
#define RENDER_PROCESS_PIXEL(RENDERFUN, PROCESSPIXELFUN) \
KERNEL RENDERFUN ## _device(Vector4u *outRendering, const Vector4f *ptsRay, const ITMVoxelBlock *voxelData,\
    const typename ITMVoxelBlockHash::IndexData *voxelIndex, Vector2i imgSize, Vector3f lightSource) { \
    int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);\
    if (x >= imgSize.x || y >= imgSize.y) return;\
    int locId = pixelLocId(x, y, imgSize);\
    Vector4f ptRay = ptsRay[locId];\
    PROCESSPIXELFUN(outRendering[locId], ptRay.toVector3(), ptRay.w > 0, voxelData, voxelIndex, lightSource);\
}

RENDER_PROCESS_PIXEL(renderGrey, processPixelGrey)
RENDER_PROCESS_PIXEL(renderColourFromNormal, processPixelNormal)
RENDER_PROCESS_PIXEL(renderColour, processPixelColour)

// class implementation
ITMVisualisationEngine::ITMVisualisationEngine(ITMScene *scene) : scene(scene)
{
    cudaSafeCall(hipMalloc((void**)&renderingBlockList_device, sizeof(RenderingBlock) * MAX_RENDERING_BLOCKS));
    cudaSafeCall(hipMalloc((void**)&noTotalBlocks_device, sizeof(uint)));
}

ITMVisualisationEngine::~ITMVisualisationEngine(void)
{
    cudaSafeCall(hipFree(noTotalBlocks_device));
    cudaSafeCall(hipFree(renderingBlockList_device));
}

ITMRenderState* ITMVisualisationEngine::CreateRenderState(const Vector2i & imgSize) const
{
    return new ITMRenderState(
        imgSize
        );
}

void ITMVisualisationEngine::CreateExpectedDepths(
    const ITMPose *pose, const ITMIntrinsics *intrinsics,
    ITMRenderState *renderState) const
{
    const float voxelSize = this->scene->sceneParams->voxelSize;

    Vector2i imgSize = renderState->renderingRangeImage->noDims;

    //go through list of voxel blocks, create rendering blocks storing min and max depth in that range
    const ITMHashEntry *hash_entries = this->scene->index.GetEntries();
    {
        dim3 blockSize(256);
        dim3 gridSize((int)ceil((float)SDF_LOCAL_BLOCK_NUM / (float)blockSize.x));

        cudaSafeCall(hipMemset(noTotalBlocks_device, 0, sizeof(uint)));

        projectAndSplitBlocks_device << <gridSize, blockSize >> >(
            hash_entries,
            scene->localVBA.GetVoxelBlocks(),
            pose->GetM(),
            intrinsics->projectionParamsSimple.all, imgSize, voxelSize,

            renderingBlockList_device, noTotalBlocks_device);
    }
    uint noTotalBlocks;
    cudaSafeCall(hipMemcpy(&noTotalBlocks, noTotalBlocks_device, sizeof(uint), hipMemcpyDeviceToHost));
    if (noTotalBlocks > (unsigned)MAX_RENDERING_BLOCKS) noTotalBlocks = MAX_RENDERING_BLOCKS;

    // go through rendering blocks and fill minmaxData
    Vector2f * const minmaxData = renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA);
    // 1. reset
    memsetKernel<Vector2f>(minmaxData, Vector2f(FAR_AWAY, VERY_CLOSE), renderState->renderingRangeImage->dataSize);

    // 2. copy from rendering blocks
    dim3 blockSize(16, 16);
    dim3 gridSize((unsigned int)ceil((float)noTotalBlocks / 4.0f), 4);
    fillBlocks_device << <gridSize, blockSize >> >(noTotalBlocks_device, renderingBlockList_device, imgSize, minmaxData);
}

/// uses renderingRangeImage, creates raycastResult
static void GenericRaycast(
    const ITMScene *const scene,
    const Vector2i& imgSize,
    const Matrix4f& invM,
    const Vector4f projParams, 
    ITMRenderState *const renderState //!< [in, out] uses renderingRangeImage, creates raycastResult
    )
{
    const float voxelSize = scene->sceneParams->voxelSize;
    const float oneOverVoxelSize = 1.0f / voxelSize;

    // for speedup (?)
    Vector4f invProjParams(1.0f / projParams.x, 1.0f / projParams.y, projParams.z, projParams.w);

    dim3 cudaBlockSize(16, 12);
    dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
    genericRaycast_device << <gridSize, cudaBlockSize >> >(
        renderState->raycastResult->GetData(MEMORYDEVICE_CUDA),
        scene->localVBA.GetVoxelBlocks(),
        scene->index.GetEntries(),
        imgSize,
        invM,
        invProjParams,
        oneOverVoxelSize,
        renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA),
        scene->sceneParams->mu
        );
}

static void RenderImage_common(
    const ITMScene *const scene,
    const ITMPose *const pose,
    const ITMIntrinsics *const intrinsics,
    ITMRenderState *const renderState,
    ITMUChar4Image *const outputImage,
    const ITMVisualisationEngine::RenderImageType type)
{
    Vector2i imgSize = outputImage->noDims;
    Matrix4f invM = pose->GetInvM();

    GenericRaycast(scene, imgSize, invM, intrinsics->projectionParamsSimple.all, renderState);

    Vector3f lightSource = -Vector3f(invM.getColumn(2));
    Vector4u *outRendering = outputImage->GetData(MEMORYDEVICE_CUDA);
    Vector4f *pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);

    dim3 cudaBlockSize(8, 8);
    dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));

    switch (type) {
    case ITMVisualisationEngine::RENDER_COLOUR_FROM_VOLUME:
        renderColour_device << <gridSize, cudaBlockSize >> >(outRendering, pointsRay, scene->localVBA.GetVoxelBlocks(),
            scene->index.GetEntries(), imgSize, lightSource);
        break;
    case ITMVisualisationEngine::RENDER_COLOUR_FROM_NORMAL:
        renderColourFromNormal_device << <gridSize, cudaBlockSize >> >(outRendering, pointsRay, scene->localVBA.GetVoxelBlocks(),
            scene->index.GetEntries(), imgSize, lightSource);
        break;
    case ITMVisualisationEngine::RENDER_SHADED_GREYSCALE:
    default:
        renderGrey_device << <gridSize, cudaBlockSize >> >(outRendering, pointsRay, scene->localVBA.GetVoxelBlocks(),
            scene->index.GetEntries(), imgSize, lightSource);
        break;
    }
}

void CreateICPMaps_common(const ITMScene *scene, Vector4f intrinsics_d, ITMTrackingState *trackingState, ITMRenderState *renderState)
{
    Vector2i imgSize = renderState->raycastResult->noDims;
    Matrix4f invM = trackingState->pose_d->GetInvM();

    GenericRaycast(scene, imgSize, invM, intrinsics_d, renderState);

    // Remember the pose from which this point cloud was rendered
    trackingState->pointCloud->pose_pointCloud->SetFrom(trackingState->pose_d);

    Vector4f *pointsMap = trackingState->pointCloud->locations->GetData(MEMORYDEVICE_CUDA);
    Vector4f *normalsMap = trackingState->pointCloud->normals->GetData(MEMORYDEVICE_CUDA);
    Vector4f *pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);

    Vector3f lightSource = -Vector3f(invM.getColumn(2));

    dim3 cudaBlockSize(16, 12);
    dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
    renderICP_device << <gridSize, cudaBlockSize >> >(pointsMap, normalsMap, pointsRay,
        scene->sceneParams->voxelSize, imgSize, lightSource);
}

void ITMVisualisationEngine::RenderImage(const ITMPose *pose, const ITMIntrinsics *intrinsics,
    ITMRenderState *renderState, ITMUChar4Image *outputImage, ITMVisualisationEngine::RenderImageType type) const
{
    CreateExpectedDepths(pose, intrinsics, renderState);
    RenderImage_common(this->scene, pose, intrinsics, renderState, outputImage, type);
}

void ITMVisualisationEngine::CreateICPMaps(
    const ITMIntrinsics * const intrinsics_d,
    ITMTrackingState *const trackingState,
    ITMRenderState *const renderStateTemp) const
{
    CreateExpectedDepths(trackingState->pose_d, intrinsics_d, renderStateTemp);
    CreateICPMaps_common(this->scene, intrinsics_d->projectionParamsSimple.all, trackingState, renderStateTemp);
}


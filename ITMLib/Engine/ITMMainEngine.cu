// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMMainEngine.h"

using namespace ITMLib::Engine;

ITMMainEngine::ITMMainEngine(const ITMLibSettings *settings, const ITMRGBDCalib *calib, Vector2i imgSize_rgb, Vector2i imgSize_d)
{
	this->settings = settings;

	scene = new ITMScene(&(settings->sceneParams));

	lowLevelEngine = new ITMLowLevelEngine();
	viewBuilder = new ITMViewBuilder(calib);
	visualisationEngine = new ITMVisualisationEngine(scene);

    Vector2i trackedImageSize = imgSize_d;

	renderState_live = visualisationEngine->CreateRenderState(trackedImageSize);
	renderState_freeview = NULL; //will be created by the visualisation engine on demand

    sceneRecoEngine = new ITMSceneReconstructionEngine();
    ResetScene();

    tracker = new ITMDepthTracker(
        trackedImageSize,
        settings->depthTrackerICPThreshold,
        settings->depthTrackerTerminationThreshold,
        lowLevelEngine
        );
    trackingState = tracker->BuildTrackingState();

	view = NULL; // will be allocated by the view builder

	fusionActive = true;
	mainProcessingActive = true;
}

ITMMainEngine::~ITMMainEngine()
{
	delete renderState_live;
    delete renderState_freeview;

	delete scene;

    delete sceneRecoEngine;

	delete tracker;

	delete lowLevelEngine;
	delete viewBuilder;

	delete trackingState;
	if (view != NULL) delete view;

	delete visualisationEngine;
}

#include <fstream>
#include <map>
void ITMMainEngine::ProcessFrame(ITMUChar4Image *rgbImage, ITMShortImage *rawDepthImage)
{
	// prepare image and turn it into a depth image
	viewBuilder->UpdateView(&view, rgbImage, rawDepthImage);

	// tracking
    tracker->TrackCamera(trackingState, view);

	// fusion
    sceneRecoEngine->ProcessFrame(view, trackingState, scene);
    if (0)
    {
        // create scene from dump [[
         std::map<VoxelBlockPos, ITMVoxelBlock> vbs; 
        {
            std::ifstream in("scenedump1.txt");
            while (1) {
                ITMVoxelBlock b;
                in >> b.pos.x >> b.pos.y >> b.pos.z;
                if (in.fail()) break;

                int i = 0;
                for (auto& v : b.blockVoxels) {
                    float s; in >> s;
                    v.setSDF(s);
                }

                vbs[b.pos] = b;
            }

            in.close();
        }

    {
        std::ifstream in("scenehashdump1.txt");
        while (1) {
            int j;
            ITMHashEntry he;
            in >> j >> he.pos.x >> he.pos.y >> he.pos.z >> he.offset >> he.ptr;
            if (in.fail()) break;

            hipMemcpy(scene->index.GetEntries() + j, &he,
                sizeof(ITMHashEntry), hipMemcpyHostToDevice);


            hipMemcpy(scene->localVBA.GetVoxelBlocks() + he.ptr, &vbs[he.pos],
                sizeof(ITMVoxelBlock), hipMemcpyHostToDevice);
        }
        in.close();
    }

    }
    // ]]


    // raycast scene from current viewpoint 
    // to create point cloud for tracking
    visualisationEngine->CreateICPMaps(&view->calib->intrinsics_d, trackingState, renderState_live);
}
void ITMMainEngine::GetImage(
    ITMUChar4Image * const out,
    const GetImageType getImageType, 
    const ITMPose * const pose, 
    const ITMIntrinsics * const intrinsics 
    )
{
    assert(out->isAllocated_CPU() && out->isAllocated_CUDA());
	if (view == NULL) return;

	out->Clear();

	switch (getImageType)
	{
	case ITMMainEngine::InfiniTAM_IMAGE_ORIGINAL_RGB:
		out->ChangeDims(view->rgb->noDims);
        out->SetFrom(view->rgb, ORUtils::MemoryBlock<Vector4u>::CUDA_TO_CPU);
        break;

	case ITMMainEngine::InfiniTAM_IMAGE_ORIGINAL_DEPTH:
		out->ChangeDims(view->depth->noDims);
        view->depth->UpdateHostFromDevice();
        ITMVisualisationEngine::DepthToUchar4(out, view->depth);
		break;

	case ITMMainEngine::InfiniTAM_IMAGE_FREECAMERA_SHADED:
	case ITMMainEngine::InfiniTAM_IMAGE_FREECAMERA_COLOUR_FROM_VOLUME:
	case ITMMainEngine::InfiniTAM_IMAGE_FREECAMERA_COLOUR_FROM_NORMAL:
	{
		ITMVisualisationEngine::RenderImageType type = ITMVisualisationEngine::RENDER_SHADED_GREYSCALE;
		if (getImageType == ITMMainEngine::InfiniTAM_IMAGE_FREECAMERA_COLOUR_FROM_VOLUME) 
            type = ITMVisualisationEngine::RENDER_COLOUR_FROM_VOLUME;
		else if (getImageType == ITMMainEngine::InfiniTAM_IMAGE_FREECAMERA_COLOUR_FROM_NORMAL) 
            type = ITMVisualisationEngine::RENDER_COLOUR_FROM_NORMAL;

		if (renderState_freeview == NULL)
            renderState_freeview = visualisationEngine->CreateRenderState(out->noDims);

        assert(renderState_freeview->raycastResult->noDims == out->noDims);

        if (0) {
            // dump hash index:
            {
                std::ofstream of("scenehashdump1.txt");

                ITMHashEntry *hes = (ITMHashEntry *)malloc(SDF_GLOBAL_BLOCK_NUM * sizeof(ITMHashEntry));

                hipMemcpy(hes,
                    scene->index.GetEntries(),
                    SDF_GLOBAL_BLOCK_NUM * sizeof(ITMHashEntry),
                    hipMemcpyDeviceToHost);


                for (int j = SDF_GLOBAL_BLOCK_NUM - 1; j >= 0; j--) {
                    ITMHashEntry& he = hes[j];
                    if (!he.isAllocated()) continue;

                    of << j << " " << he.pos.x << " " << he.pos.y << " " << he.pos.z << " " << he.offset << " " << he.ptr << std::endl;
                }
                of.close();
                exit(0);

                exit(0);
            }
            // dump scene [[
            std::ofstream of("scenedump1.txt");
            for (int j = SDF_LOCAL_BLOCK_NUM - 1; j >= 0; j--) {
                ITMVoxelBlock b;
                hipMemcpy(&b, scene->localVBA.GetVoxelBlocks() + j,
                    sizeof(ITMVoxelBlock), hipMemcpyDeviceToHost);
                if (b.pos == INVALID_VOXEL_BLOCK_POS) continue;

                of << b.pos.x << " " << b.pos.y << " " << b.pos.z << " " << std::endl;
                int i = 0;
                for (auto& v : b.blockVoxels) {
                    of << v.getSDF() << " ";
                }
                of << std::endl;
            }
            of.close();
            exit(0);
            // ]]
        }

		visualisationEngine->RenderImage(pose, intrinsics, renderState_freeview, out, type);
        out->UpdateHostFromDevice();
		break;
	}
	case ITMMainEngine::InfiniTAM_IMAGE_UNKNOWN:
		break;
	};
}
#include "hip/hip_runtime.h"
#include "ITMSceneReconstructionEngine.h"
#include "ITMCUDAUtils.h"
#include "ITMLibDefines.h"
#include "ITMPixelUtils.h"
#include "ITMRepresentationAccess.h"
#include "CoordinateSystem.h"
#include "CameraImage.h"

/// Fusion Stage - Camera Data Integration
/// \returns \f$\eta\f$, -1 on failure
// Note that the stored T-SDF values are normalized to lie
// in [-1,1] within the truncation band.
GPU_ONLY inline float computeUpdatedVoxelDepthInfo(
    DEVICEPTR(ITMVoxel) &voxel, //!< X
    const THREADPTR(Point) & pt_model //!< in world space
    )
{

    // project point into depth image
    /// X_d, depth camera coordinate system
    const Vector4f pt_camera = Vector4f(
        currentView->depthImage->eyeCoordinates->convert(pt_model).location,
        1);
    /// \pi(K_dX_d), projection into the depth image
    Vector2f pt_image;
    if (!currentView->depthImage->project(pt_model, pt_image))
        return -1;

    // get measured depth from image, no interpolation
    /// I_d(\pi(K_dX_d))
    auto p = currentView->depthImage->getPointForPixel(pt_image.toInt());
    const float depth_measure = p.location.z;
    if (depth_measure <= 0.0) return -1;

    /// I_d(\pi(K_dX_d)) - X_d^(z)          (3)
    float const eta = depth_measure - pt_camera.z;
    // check whether voxel needs updating
    if (eta < -mu) return eta;

    // compute updated SDF value and reliability (number of observations)
    /// D(X), w(X)
    float const oldF = voxel.getSDF();
    int const oldW = voxel.w_depth;

    // newF, normalized for -1 to 1
    float const newF = MIN(1.0f, eta / mu);
    int const newW = 1;

    updateVoxelDepthInformation(
        voxel,
        oldF, oldW, newF, newW);

    return eta;
}

/// \returns early on failure
GPU_ONLY inline void computeUpdatedVoxelColorInfo(
    DEVICEPTR(ITMVoxel) &voxel,
    const THREADPTR(Point) & pt_model)
{
    Vector2f pt_image;
    if (!currentView->colorImage->project(pt_model, pt_image))
        return;

    int oldW = (float)voxel.w_color;
    const Vector3f oldC = TO_FLOAT3(voxel.clr);

    /// Like formula (4) for depth
    const Vector3f newC = TO_VECTOR3(interpolateBilinear<Vector4f>(currentView->colorImage->image->GetData(), pt_image, currentView->colorImage->imgSize()));
    int newW = 1;

    updateVoxelColorInformation(
        voxel,
        oldC, oldW, newC, newW);
}


GPU_ONLY static void computeUpdatedVoxelInfo(
    DEVICEPTR(ITMVoxel) & voxel, //!< [in, out] updated voxel
    const THREADPTR(Point) & pt_model) {
    const float eta = computeUpdatedVoxelDepthInfo(voxel, pt_model);

    // Only the voxels within +- 25% mu of the surface get color
    if ((eta > mu) || (fabs(eta / mu) > 0.25f)) return;
    computeUpdatedVoxelColorInfo(voxel, pt_model);
}

/// Determine the blocks around a given depth sample that are currently visible
/// and need to be allocated.
/// Builds hashVisibility and entriesAllocType.
/// \param x,y [in] loop over depth image.
struct buildHashAllocAndVisibleTypePP {
    forEachPixelNoImage_process() {
        // Find 3d position of depth pixel xy, in eye coordinates
        auto pt_camera = currentView->depthImage->getPointForPixel(Vector2i(x, y));

        const float depth = pt_camera.location.z;
        if (depth <= 0 || (depth - mu) < 0 || (depth - mu) < viewFrustum_min || (depth + mu) > viewFrustum_max) return;

        // the found point +- mu
        const Vector pt_camera_v = (pt_camera - currentView->depthImage->location());
        const float norm = length(pt_camera_v.direction);
        const Vector pt_camera_v_minus_mu = pt_camera_v*(1.0f - mu / norm);
        const Vector pt_camera_v_plus_mu = pt_camera_v*(1.0f + mu / norm);

        // Convert to voxel block coordinates  
        // the initial point pt_camera_v_minus_mu
        Point point = voxelBlockCoordinates->convert(currentView->depthImage->location() + pt_camera_v_minus_mu);
        // the direction towards pt_camera_v_plus_mu in voxelBlockCoordinates
        const Vector vector = voxelBlockCoordinates->convert(pt_camera_v_plus_mu - pt_camera_v_minus_mu);

        // We will step along point -> point_e and add all voxel blocks we encounter to the visible list
        // "Create a segment on the line of sight in the range of the T-SDF truncation band"
        const int noSteps = (int)ceil(2.0f* length(vector.direction) ); // make steps smaller than 1, maybe even < 1/2 to really land in all blocks at least once
        const Vector direction = vector * (1.f / (float)(noSteps - 1));

        //add neighbouring blocks
        for (int i = 0; i < noSteps; i++)
        {
            // "take the block coordinates of voxels on this line segment"
            const VoxelBlockPos blockPos = TO_SHORT_FLOOR3(point.location);
            Scene::requestCurrentSceneVoxelBlockAllocation(blockPos);

            point = point + direction;
        }
    }
};

#include <hip/hip_runtime.h>

struct IntegrateVoxel {
    doForEachAllocatedVoxel_process() {
        computeUpdatedVoxelInfo(*v, globalPoint);
    }
};

/// Fusion stage of the system
void Fuse()
{
    hipDeviceSynchronize();
    assert(Scene::getCurrentScene());
    assert(currentView);

    // allocation request
    forEachPixelNoImage<buildHashAllocAndVisibleTypePP>(currentView->depthImage->imgSize());
    hipDeviceSynchronize();

    // allocation
    Scene::performCurrentSceneAllocations();

    // camera data integration
    hipDeviceSynchronize();
    Scene::getCurrentScene()->doForEachAllocatedVoxel<IntegrateVoxel>();
}
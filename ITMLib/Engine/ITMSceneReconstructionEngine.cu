#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSceneReconstructionEngine.h"
#include "ITMCUDAUtils.h"
#include "ITMLibDefines.h"
#include "ITMPixelUtils.h"
#include "ITMRepresentationAccess.h"
#include "ITMLocalVBA.h"

using namespace ITMLib::Engine;



/// Fusion Stage - Camera Data Integration
/// \returns \f$\eta\f$, -1 on failure
// Note that the stored T-SDF values are normalized to lie
// in [-1,1] within the truncation band.
CPU_AND_GPU inline float computeUpdatedVoxelDepthInfo(
    DEVICEPTR(ITMVoxel) &voxel, //!< X
    const THREADPTR(Vector4f) & pt_model, //!< voxel location X
    const CONSTPTR(Matrix4f) & M_d, //!< depth camera pose
    const CONSTPTR(Vector4f) & projParams_d, //!< intrinsic camera parameters \f$K_d\f$
    float mu, int maxW, const CONSTPTR(float) *depth, const CONSTPTR(Vector2i) & imgSize)
{

    float depth_measure, eta, oldF, newF;
    int oldW, newW;

    // project point into depth image
    /// X_d, depth camera coordinate system
    Vector4f pt_camera;
    /// \pi(K_dX_d), projection into the depth image
    Vector2f pt_image;
    if (!projectModel(projParams_d, M_d,
        imgSize, pt_model, pt_camera, pt_image)) return -1;

    // get measured depth from image, no interpolation
    /// I_d(\pi(K_dX_d))
    depth_measure = sampleNearest(depth, pt_image, imgSize);
    if (depth_measure <= 0.0) return -1;

    /// I_d(\pi(K_dX_d)) - X_d^(z)          (3)
    eta = depth_measure - pt_camera.z;
    // check whether voxel needs updating
    if (eta < -mu) return eta;

    // compute updated SDF value and reliability (number of observations)
    /// D(X), w(X)
    oldF = voxel.getSDF();
    oldW = voxel.w_depth;

    // newF, normalized for -1 to 1
    newF = MIN(1.0f, eta / mu);
    newW = 1;

    updateVoxelDepthInformation(
        voxel,
        oldF, oldW, newF, newW, maxW);

    return eta;
}

/// \returns early on failure
CPU_AND_GPU inline void computeUpdatedVoxelColorInfo(DEVICEPTR(ITMVoxel) &voxel, const THREADPTR(Vector4f) & pt_model, const CONSTPTR(Matrix4f) & M_rgb,
    const CONSTPTR(Vector4f) & projParams_rgb, float mu, uchar maxW, float eta, const CONSTPTR(Vector4u) *rgb, const CONSTPTR(Vector2i) & imgSize)
{
    Vector4f pt_camera; Vector2f pt_image;
    Vector3f oldC, newC;
    int newW, oldW;

    if (!projectModel(projParams_rgb, M_rgb,
        imgSize, pt_model, pt_camera, pt_image)) return;

    oldW = (float)voxel.w_color;
    oldC = TO_FLOAT3(voxel.clr);

    /// Like formula (4) for depth
    newC = TO_VECTOR3(interpolateBilinear<Vector4f>(rgb, pt_image, imgSize));
    newW = 1;

    updateVoxelColorInformation(
        voxel,
        oldC, oldW, newC, newW, maxW);
}


CPU_AND_GPU static void computeUpdatedVoxelInfo(
    DEVICEPTR(ITMVoxel) & voxel, //!< [in, out] updated voxel
    const THREADPTR(Vector4f) & pt_model,
    const THREADPTR(Matrix4f) & M_d, const THREADPTR(Vector4f) & projParams_d,
    const THREADPTR(Matrix4f) & M_rgb, const THREADPTR(Vector4f) & projParams_rgb,
    float mu, int maxW,
    const CONSTPTR(float) *depth, const CONSTPTR(Vector2i) & imgSize_d,
    const CONSTPTR(Vector4u) *rgb, const THREADPTR(Vector2i) & imgSize_rgb)
{
    float eta = computeUpdatedVoxelDepthInfo(voxel, pt_model, M_d, projParams_d, mu, maxW, depth, imgSize_d);

    // Only the voxels withing +- 25% mu of the surface get color
    if ((eta > mu) || (fabs(eta / mu) > 0.25f)) return;
    computeUpdatedVoxelColorInfo(voxel, pt_model, M_rgb, projParams_rgb, mu, maxW, eta, rgb, imgSize_rgb);
}

// alloc types
#define AT_NEEDS_ALLOC_FITS 1 //needs allocation, fits in the ordered list
#define AT_NEEDS_ALLOC_EXCESS 2 //needs allocation in the excess list

/// For allocation and visibility determination. 
///
/// Determine the blocks around a given depth sample that are currently visible
/// and need to be allocated.
/// Builds hashVisibility and entriesAllocType.
/// \param x,y [in] loop over depth image.
CPU_AND_GPU inline void buildHashAllocAndVisibleTypePP(
    DEVICEPTR(uchar) * const entriesAllocType, //!< [out] allocation type (AT_*) for each hash table bucket, indexed by values computed from hashIndex, or in excess part
    const int x, const int y,
    DEVICEPTR(Vector4s) * const  blockCoords, //!< [out] blockPos coordinate of each voxel block that needs allocation, indexed by values computed from hashIndex, or in excess part
    const CONSTPTR(float) * const  depth,
    const Matrix4f invM_d, //!< depth to world transformation
    const Vector4f invProjParams_d, //!< Note: Inverse projection parameters to avoid division by fx, fy.
    const float mu,
    const Vector2i imgSize,
    const float oneOverVoxelBlockWorldspaceSize, //!< 1 / (voxelSize * SDF_BLOCK_SIZE)
    const CONSTPTR(ITMHashEntry) *const hashTable, //<! [in] hash table buckets, indexed by values computed from hashIndex
    const float viewFrustum_min, //!< znear
    const float viewFrustum_max  //!< zfar
    )
{
    float depth_measure; unsigned int hashIdx; int noSteps;
    Vector4f pt_camera_f; Vector3f point_e, point, direction; Vector3s blockPos;

    // Find 3d position of depth pixel xy
    depth_measure = depth[x + y * imgSize.x];
    if (depth_measure <= 0 || (depth_measure - mu) < 0 || (depth_measure - mu) < viewFrustum_min || (depth_measure + mu) > viewFrustum_max) return;

    pt_camera_f = depthTo3DInvProjParams(invProjParams_d, x, y, depth_measure);

    // distance from camera
    float norm = length(pt_camera_f.toVector3());

    // Transform into block coordinates the found point +- mu
    // TODO why /norm? An adhoc fix to not allocate too much when far away and allocate more when nearby?
    point = TO_VECTOR3(invM_d * (pt_camera_f * (1.0f - mu / norm))) * oneOverVoxelBlockWorldspaceSize;
    point_e = TO_VECTOR3(invM_d * (pt_camera_f * (1.0f + mu / norm))) * oneOverVoxelBlockWorldspaceSize;

    // We will step along point -> point_e and add all voxel blocks we encounter to the visible list
    // "Create a segment on the line of sight in the range of the T-SDF truncation band"
    direction = point_e - point;
    norm = length(direction);
    noSteps = (int)ceil(2.0f*norm);

    direction /= (float)(noSteps - 1);

    //add neighbouring blocks
    for (int i = 0; i < noSteps; i++)
    {
        // "take the block coordinates of voxels on this line segment"
        blockPos = TO_SHORT_FLOOR3(point);

        //compute index in hash table
        hashIdx = hashIndex(blockPos);

        //check if hash table contains entry (block has already been allocated)
        bool isFound = false;

        ITMHashEntry hashEntry;

        // whether we find blockPos at the current hashIdx
#define check_found(BREAK) \
            hashEntry = hashTable[hashIdx]; \
            if (IS_EQUAL3(hashEntry.pos, blockPos) && hashEntry.isAllocated()) \
                        {\
                isFound = true; \
                BREAK;\
                        }

        check_found(NULL);

        if (!isFound)
        {
            bool isExcess = false;
            if (hashEntry.isAllocated()) //seach excess list only if there is no room in ordered part
            {
                isExcess = true;
                while (hashEntry.hasExcessListOffset())
                {
                    hashIdx = hashEntry.getHashIndexOfNextExcessEntry();
                    check_found(break);
                }
            }

            if (!isFound) //still not found: needs allocation 
            {
                entriesAllocType[hashIdx] = isExcess ? AT_NEEDS_ALLOC_EXCESS : AT_NEEDS_ALLOC_FITS; //needs allocation
                blockCoords[hashIdx] = Vector4s(blockPos, 1);
            }
        }

        point += direction;
    }
#undef check_found
}

#include <hip/hip_runtime.h>

/// \returns false when the list is full

inline
__device__
void allocateVoxelBlock(
int targetIdx,

typename ITMLocalVBA::VoxelAllocationList* voxelAllocationList,
ITMVoxelBlockHash::ExcessAllocationList* excessAllocationList,
ITMHashEntry *hashTable,
ITMVoxelBlock *localVBA,

uchar *entriesAllocType,
Vector4s *blockCoords)
{
    const unsigned char hashChangeType = entriesAllocType[targetIdx];
    if (hashChangeType == 0) return;
    const int ptr = voxelAllocationList->Allocate();
    assert(ptr >= 0 && ptr < SDF_LOCAL_BLOCK_NUM);
    //if (ptr < 0) return; //there is no room in the voxel block array


    ITMHashEntry hashEntry;
    hashEntry.pos = TO_SHORT3(blockCoords[targetIdx]);
    hashEntry.ptr = ptr;
    hashEntry.offset = 0;

    // Allocated voxel block - back-reference to key:
    assert(localVBA[ptr].pos == INVALID_VOXEL_BLOCK_POS); // make sure this was free before
    localVBA[ptr].pos = hashEntry.pos;

    if (hashChangeType == AT_NEEDS_ALLOC_EXCESS) { //needs allocation in the excess list
        const int exlOffset = excessAllocationList->Allocate();

        if (exlOffset >= 0) //there is room in the excess list
        {
            hashTable[targetIdx].offset = exlOffset + 1; //connect parent to child

            targetIdx = SDF_BUCKET_NUM + exlOffset; // target index is in excess part
        }
    }

    hashTable[targetIdx] = hashEntry;
}


CPU_AND_GPU inline void integrateVoxel(int x, int y, int z,
    Vector3i globalPos,
    ITMVoxelBlock *localVoxelBlock,
    float voxelSize,

    const CONSTPTR(Matrix4f) & M_d, const CONSTPTR(Vector4f) & projParams_d,
    const CONSTPTR(Matrix4f) & M_rgb, const CONSTPTR(Vector4f) & projParams_rgb,
    float mu, int maxW,
    const CONSTPTR(float) *depth, const CONSTPTR(Vector2i) & depthImgSize,
    const CONSTPTR(Vector4u) *rgb, const CONSTPTR(Vector2i) & rgbImgSize
    ) {
    const int locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

    // Voxel's world coordinates, for later projection into depth and color image
    const Vector4f pt_model = Vector4f(
        (globalPos.toFloat() + Vector3f((float)x, (float)y, (float)z)) * voxelSize, 1.f);

    computeUpdatedVoxelInfo(
        localVoxelBlock->blockVoxels[locId],
        pt_model,
        M_d,
        projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

// device functions
KERNEL integrateIntoScene_device(
    ITMVoxelBlock *localVBA, //!< [out]

    const ITMHashEntry * const hashTable,
    const Vector4u * const rgb,
    const Vector2i rgbImgSize, 
    const float * const depth,
    const Vector2i depthImgSize, const Matrix4f M_d, const Matrix4f M_rgb, const Vector4f projParams_d,
    const Vector4f projParams_rgb, const float voxelSize, const float mu, const int maxW)
{
    // one thread block for each voxel block
    ITMVoxelBlock * const localVoxelBlock = &(localVBA[blockIdx.x]); 
    if (localVoxelBlock->pos == INVALID_VOXEL_BLOCK_POS) return; 

    // one thread for each voxel
    const int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;
    integrateVoxel(x, y, z,
        localVoxelBlock->pos.toInt() * SDF_BLOCK_SIZE,
        localVoxelBlock, voxelSize,
        M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

KERNEL buildHashAllocAndVisibleType_device(
    uchar *entriesAllocType, //!< [out]
    Vector4s *blockCoords,//!< [out]
    const float *depth,
    Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i imgSize, float voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
    float viewFrustum_max)
{
    const int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;

    buildHashAllocAndVisibleTypePP(entriesAllocType, x, y, blockCoords, depth, invM_d,
        projParams_d, mu, imgSize, voxelSize, hashTable, viewFrustum_min, viewFrustum_max);
}


KERNEL allocateVoxelBlocksList_device(
    typename ITMLocalVBA::VoxelAllocationList * const voxelAllocationList,
    ITMVoxelBlockHash::ExcessAllocationList * const excessAllocationList,
    ITMHashEntry *hashTable,
    ITMVoxelBlock* localVBA,
    uchar *entriesAllocType,
    Vector4s *blockCoords)
{
    const int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (targetIdx > SDF_GLOBAL_BLOCK_NUM - 1) return;

    allocateVoxelBlock(targetIdx,
        voxelAllocationList,
        excessAllocationList,
        hashTable,
        localVBA,

        entriesAllocType,
        blockCoords);
}


// host methods

ITMSceneReconstructionEngine::ITMSceneReconstructionEngine(void) 
{
    cudaSafeCall(hipMalloc((void**)&entriesAllocType_device, SDF_GLOBAL_BLOCK_NUM));
    cudaSafeCall(hipMalloc((void**)&blockCoords_device, SDF_GLOBAL_BLOCK_NUM * sizeof(Vector4s)));
}

ITMSceneReconstructionEngine::~ITMSceneReconstructionEngine(void) 
{
	cudaSafeCall(hipFree(entriesAllocType_device));
	cudaSafeCall(hipFree(blockCoords_device));
}

/// thread blocks 0:numBlocks-1, threads 0:SDF_BLOCK_SIZE3-1
KERNEL resetVoxelBlocks(ITMVoxelBlock *voxelBlocks_ptr) {
    voxelBlocks_ptr[blockIdx.x].blockVoxels[threadIdx.x] = ITMVoxel();

    if (threadIdx.x == 0) voxelBlocks_ptr[blockIdx.x].pos = INVALID_VOXEL_BLOCK_POS;
}

void ITMSceneReconstructionEngine::ResetScene(ITMScene *scene)
{
    printf("ResetScene\n");

    // Reset sdf data of all voxels in all voxel blocks
    ITMVoxelBlock *voxelBlocks_ptr = scene->localVBA.GetVoxelBlocks();
    resetVoxelBlocks << <SDF_LOCAL_BLOCK_NUM, SDF_BLOCK_SIZE3 >> >(voxelBlocks_ptr);

    // Reset voxel allocation list
    scene->localVBA.voxelAllocationList->Reset();

    // Reset hash entries
    memsetKernel<ITMHashEntry>(
        scene->index.GetEntries(),
        ITMHashEntry::createIllegalEntry(),
        SDF_GLOBAL_BLOCK_NUM);

    // Reset excess allocation list
    scene->index.excessAllocationList->Reset();
    printf("ResetScene done\n");
}

void ITMSceneReconstructionEngine::AllocateSceneFromDepth(
    ITMScene *scene,
    const ITMView *view, 
	const ITMTrackingState *trackingState)
{
	const Vector2i depthImgSize = view->depth->noDims;
    const float voxelSize = scene->sceneParams->voxelSize;
    
    const Matrix4f M_d = trackingState->pose_d->GetM();
    const Matrix4f invM_d = trackingState->pose_d->GetInvM();

    const Vector4f invProjParams_d = view->calib->intrinsics_d.getInverseProjParams();

	const float mu = scene->sceneParams->mu;

    float * const depth = view->depth->GetData(MEMORYDEVICE_CUDA);
    auto voxelAllocationList = scene->localVBA.voxelAllocationList;
    auto excessAllocationList = scene->index.excessAllocationList;
    ITMHashEntry * const hashTable = scene->index.GetEntries();


    const dim3 cudaBlockSizeHV(16, 16);
    const dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	const dim3 cudaBlockSizeAL(256, 1);
	const dim3 gridSizeAL((int)ceil((float)SDF_GLOBAL_BLOCK_NUM / (float)cudaBlockSizeAL.x));

    const float oneOverVoxelSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);


    // Determine blocks currently visible in depth map but not allocated  for preparing allocation list

    cudaSafeCall(hipMemsetAsync(entriesAllocType_device, 0, sizeof(unsigned char)* SDF_GLOBAL_BLOCK_NUM));

	buildHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(
        entriesAllocType_device,
		blockCoords_device,
        depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min,
        scene->sceneParams->viewFrustum_max);

    // Do allocation
    allocateVoxelBlocksList_device << <gridSizeAL, cudaBlockSizeAL >> >(
        voxelAllocationList, 
        excessAllocationList, 
        hashTable,
        scene->localVBA.GetVoxelBlocks(),
        entriesAllocType_device, 
		blockCoords_device);
}

void ITMSceneReconstructionEngine::IntegrateIntoScene(ITMScene *scene, const ITMView *view,
	const ITMTrackingState *trackingState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	M_d = trackingState->pose_d->GetM();
    M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * M_d;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
    ITMVoxelBlock *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
    dim3 gridSize(SDF_LOCAL_BLOCK_NUM);

    integrateIntoScene_device<< <gridSize, cudaBlockSize >> >(
        localVBA, hashTable, 
        rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
}

